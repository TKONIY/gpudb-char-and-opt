#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <chrono>
#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <unistd.h>

#include "cub/test/test_util.h"
#include <cub/util_allocator.cuh>
#include <hip/hip_runtime.h>

#include "crystal/crystal.cuh"

#include "utils/generator.h"
#include "utils/gpu_utils.h"

using namespace std;

#define DEBUG 1

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_kernel(int *dim_key, int *dim_val, int num_tuples,
                             int *hash_table, int num_slots) {
  int items[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = num_tuples - tile_offset;
  }

  InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items,
                                                  num_tile_items);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2,
                                                  num_tile_items);
  BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      items, items2, selection_flags, hash_table, num_slots, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void probe_kernel(int *fact_fkey, int *fact_val, int num_tuples,
                             int *hash_table, int num_slots,
                             unsigned long long *res) {
  // Load a tile striped across threads
  int selection_flags[ITEMS_PER_THREAD];
  int keys[ITEMS_PER_THREAD];
  int vals[ITEMS_PER_THREAD];
  int join_vals[ITEMS_PER_THREAD];

  unsigned long long sum = 0;

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = num_tuples - tile_offset;
  }

  InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(fact_fkey + tile_offset, keys,
                                                  num_tile_items);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(fact_val + tile_offset, vals,
                                                  num_tile_items);

  BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      keys, join_vals, selection_flags, hash_table, num_slots, num_tile_items);

#pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
    if ((threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items))
      if (selection_flags[ITEM])
        sum += vals[ITEM] * join_vals[ITEM];
  }

  __syncthreads();

  static __shared__ long long buffer[32];
  unsigned long long aggregate =
      BlockSum<long long, BLOCK_THREADS, ITEMS_PER_THREAD>(sum,
                                                           (long long *)buffer);
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(res, aggregate);
  }
}

struct TimeKeeper {
  float time_build;
  float time_probe;
  float time_extra;
  float time_total;
};

TimeKeeper hashJoin(int *d_dim_key, int *d_dim_val, int *d_fact_fkey,
                    int *d_fact_val, int num_dim, int num_fact,
                    hipcub::CachingDeviceAllocator &g_allocator) {
  SETUP_TIMING();

  int *hash_table = NULL;
  unsigned long long *res;
  int num_slots = num_dim;
  float time_build, time_probe, time_memset, time_memset2;

  ALLOCATE(hash_table, sizeof(int) * 2 * num_dim);
  ALLOCATE(res, sizeof(long long));

  TIME_FUNC(hipMemset(hash_table, 0, num_slots * sizeof(int) * 2),
            time_memset);
  TIME_FUNC(hipMemset(res, 0, sizeof(long long)), time_memset2);

  int tile_items = 128 * 4;

  TIME_FUNC(
      (build_kernel<128, 4><<<(num_dim + tile_items - 1) / tile_items, 128>>>(
          d_dim_key, d_dim_val, num_dim, hash_table, num_slots)),
      time_build);

  auto st = chrono::high_resolution_clock::now();
  TIME_FUNC(
      (probe_kernel<128, 4><<<(num_fact + tile_items - 1) / tile_items, 128>>>(
          d_fact_fkey, d_fact_val, num_fact, hash_table, num_slots, res)),
      time_probe);
  hipDeviceSynchronize();
  auto fin = chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = fin - st;
  cout << "Execution time: " << diff.count() * 1000 << " ms" << endl;

#if DEBUG
  /* cout << "{" */
  /*      << "\"time_memset\":" << time_memset << ",\"time_build\"" << time_build */
  /*      << ",\"time_probe\":" << time_probe << "}" << endl; */
#endif

  CLEANUP(hash_table);
  CLEANUP(res);

  TimeKeeper t = {time_build, time_probe, time_memset,
                  time_build + time_probe + time_memset};
  return t;
}

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool g_verbose = false; // Whether to display input/output to console
hipcub::CachingDeviceAllocator
    g_allocator(true); // Caching allocator for device memory

#define CLEANUP(vec)                                                           \
  if (vec)                                                                     \
  CubDebugExit(g_allocator.DeviceFree(vec))

//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------
int main(int argc, char **argv) {
  using milli = chrono::milliseconds;
  auto st = chrono::high_resolution_clock::now();

  int num_fact = 256 * 1 << 20;
  int num_dim = 16 * 1 << 20;
  int num_trials = 3;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("n", num_fact);
  args.GetCmdLineArgument("d", num_dim);
  args.GetCmdLineArgument("t", num_trials);

  // Print usage
  if (args.CheckCmdLineFlag("help")) {
    printf("%s "
           "[--n=<num fact>] "
           "[--d=<num dim>] "
           "[--t=<num trials>] "
           "[--device=<device-id>] "
           "[--v] "
           "\n",
           argv[0]);
    exit(0);
  }

  int log2 = 0;
  int num_dim_dup = num_dim >> 1;
  while (num_dim_dup) {
    num_dim_dup >>= 1;
    log2 += 1;
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  int *h_dim_key = (int *)malloc(sizeof(int) * num_dim);
  int *h_dim_val = (int *)malloc(sizeof(int) * num_dim);
  int *h_fact_fkey = (int *)malloc(sizeof(int) * num_fact);
  int *h_fact_val = (int *)malloc(sizeof(int) * num_fact);

  ifstream if_dim_key("./data/storage/join_dim_" + to_string(num_dim) +
                          "_key.bin",
                      ios::in | ios::binary);
  if (!if_dim_key.good()) {
    cout << "./data/storage/join_dim_" + to_string(num_dim) +
                "_key.bin does not exist!"
         << endl;
    exit(1);
  }
  if_dim_key.read(reinterpret_cast<char *>(h_dim_key), sizeof(int) * num_dim);
  if_dim_key.close();

  ifstream if_dim_val("./data/storage/join_dim_" + to_string(num_dim) +
                          "_val.bin",
                      ios::in | ios::binary);
  if (!if_dim_val.good()) {
    cout << "./data/storage/join_dim_" + to_string(num_dim) +
                "_val.bin does not exist!"
         << endl;
    exit(1);
  }
  if_dim_val.read(reinterpret_cast<char *>(h_dim_val), sizeof(int) * num_dim);
  if_dim_val.close();

  ifstream if_fact_key("./data/storage/join_fact_" + to_string(num_fact) +
                           "_key.bin",
                       ios::in | ios::binary);
  if (!if_fact_key.good()) {
    cout << "./data/storage/join_fact_" + to_string(num_fact) +
                "_val.bin does not exist!"
         << endl;
    exit(1);
  }
  if_fact_key.read(reinterpret_cast<char *>(h_fact_fkey),
                   sizeof(int) * num_fact);
  if_fact_key.close();

  ifstream if_fact_val("./data/storage/join_fact_" + to_string(num_fact) +
                           "_val.bin",
                       ios::in | ios::binary);
  if (!if_fact_val.good()) {
    cout << "./data/storage/join_fact_" + to_string(num_fact) +
                "_val.bin does not exist!"
         << endl;
    exit(1);
  }
  if_fact_val.read(reinterpret_cast<char *>(h_fact_val),
                   sizeof(int) * num_fact);
  if_fact_val.close();

  // create_relation_pk(h_dim_key, h_dim_val, num_dim);
  // create_relation_fk(h_fact_fkey, h_fact_val, num_fact, num_dim);

  // trial counting
  int trial = 1;

  nsys_attach(trial, num_trials);

  // Allocate problem device arrays
  int *d_dim_key = NULL;
  int *d_dim_val = NULL;
  int *d_fact_fkey = NULL;
  int *d_fact_val = NULL;
  CubDebugExit(
      g_allocator.DeviceAllocate((void **)&d_dim_key, sizeof(int) * num_dim));
  CubDebugExit(
      g_allocator.DeviceAllocate((void **)&d_dim_val, sizeof(int) * num_dim));
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_fact_fkey,
                                          sizeof(int) * num_fact));
  CubDebugExit(
      g_allocator.DeviceAllocate((void **)&d_fact_val, sizeof(int) * num_fact));

  CubDebugExit(hipMemcpy(d_dim_key, h_dim_key, sizeof(int) * num_dim,
                          hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_dim_val, h_dim_val, sizeof(int) * num_dim,
                          hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_fact_fkey, h_fact_fkey, sizeof(int) * num_fact,
                          hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_fact_val, h_fact_val, sizeof(int) * num_fact,
                          hipMemcpyHostToDevice));

  TimeKeeper t = hashJoin(d_dim_key, d_dim_val, d_fact_fkey, d_fact_val,
                          num_dim, num_fact, g_allocator);

  auto fin = chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = fin - st;
  // cout << "Execution time: " << diff.count() * 1000 << " ms" << endl;

  nsys_detach(trial, num_trials);



  for (int j = 0; j < num_trials - 1; j++) {
    trial++;

    nsys_attach(trial, num_trials);
    st = chrono::high_resolution_clock::now();
    TimeKeeper t = hashJoin(d_dim_key, d_dim_val, d_fact_fkey, d_fact_val,
                            num_dim, num_fact, g_allocator);
    fin = chrono::high_resolution_clock::now();
    diff = fin - st;
    // cout << "Execution time: " << diff.count() * 1000 << " ms" << endl;
    nsys_detach(trial, num_trials);
  }

  CLEANUP(d_dim_key);
  CLEANUP(d_dim_val);
  CLEANUP(d_fact_fkey);
  CLEANUP(d_fact_val);

  return 0;
}
